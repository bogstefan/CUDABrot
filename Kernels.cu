#include "hip/hip_runtime.h"
#include "Kernels.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#define CU_SAFE_CALL(call)\
		{\
			hipError_t err = call;\
		    if( hipSuccess != err)\
			{\
		        std::cerr <<"Cuda driver error " << hipGetErrorString(err) <<" in file " << __FILE__" in line " << __LINE__ << std::endl;\
		        __debugbreak();\
		    }\
		}

__constant__
unsigned lookup[256];

__device__ __inline__
double Map(double input, double fromA, double fromB, double toA, double toB)
{
	return (input - fromA) * (toB - toA) / (fromB - fromA) + toA;
}

void SetLookup(const std::array<Color, 256>& palette)
{
	CU_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(lookup), &*palette.begin(), 256 * sizeof(unsigned int)));
}

void InitCuda(const mb::Buffer& buffer)
{
	CU_SAFE_CALL(cudaGLRegisterBufferObject(buffer.GetId()));
}

__global__
void CalculateCuda(unsigned int* pixels, Bounds bounds, ScreenDim dim, int iterations)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dim.width * dim.height) { return; }

	const int h = idx / dim.width;
	const int w = idx % dim.width;

	const double real = Map(w, 0, dim.width, bounds.left, bounds.right);
	const double imag = Map(h, 0, dim.height, bounds.down, bounds.up);

	const Complex num = { real, imag };

	Complex current = { 0,0 };
	unsigned int result = 0;

	for (int i = 0; i < iterations; ++i)
	{
		const Complex temp = current;
		current.real = temp.real * temp.real - temp.imag * temp.imag + num.real;
		current.imag = 2 * temp.real * temp.imag + num.imag;
		if (current.real * current.real + current.imag * current.imag > 4)
		{
			result = i;
			break;
		}
	}

	result = Map(result, 0, iterations, 0, 255);

	//result = result > 255 ? 100 : result;

	pixels[idx] = lookup[result];
}

void Calculate(unsigned int* pixels, Bounds bounds, ScreenDim dim, int iterations, const mb::Buffer& buffer)
{
	const int tpb = 256;
	const int blocks = dim.width * dim.height / tpb + 1;

	CU_SAFE_CALL(cudaGLMapBufferObject(reinterpret_cast<void**>(&pixels), buffer.GetId()));
	CalculateCuda KERNEL_ARGS2(blocks, tpb) (pixels, bounds, dim, iterations);
	CU_SAFE_CALL(cudaGLUnmapBufferObject(buffer.GetId()));
}
